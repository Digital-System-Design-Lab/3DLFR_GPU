#include "hip/hip_runtime.h"
#include "LF_Renderer.cuh"

// #define USE_UNIFIED_MEMORY
#define PREFETCHING_MODE 0

LF_Renderer::LF_Renderer(const std::string& path_LF, const std::string& path_pixelrange, const size_t& iw, const size_t& ih, const size_t& lf_length, const size_t& num_LFs, const double& dpp, const int& stride, const int& initPosX, const int& initPosY, bool use_window)
{
	this->stride = stride;
	size_t output_width = (size_t)(360.0 / dpp);
#ifdef USE_UNIFIED_MEMORY
	synthesized_view = alloc_uint8(output_width * ih * 3, "unified"); // output view�� ���� ����
#else
	synthesized_view = alloc_uint8(output_width * ih * 3, "device"); // output view�� ���� ����
#endif
	
	generate_LFConfig(path_LF, path_pixelrange, iw, ih, lf_length, num_LFs, dpp);
	LRU_Cache* lru_cache = new LRU_Cache(this->config, &state_h2d_thread); // LRU ĳ�� �ʱ�ȭ (LF ����, ĳ���� ���� ����)
	this->LRU = lru_cache;

	this->use_window = use_window;
	const size_t light_field_size = (this->config->LF_width * this->config->LF_height * this->config->LF_length * 3) / 2; // ����Ʈ �ʵ� ���� ������

	curPosX = initPosX; // �ʱ� ��ġ
	curPosY = initPosY;
	prevPosX = curPosX;
	prevPosY = curPosY;
	prevprevPosX = prevPosX;
	prevprevPosY = prevPosY;
	prevprevprevPosX = prevprevPosX;
	prevprevprevPosY = prevprevPosY;

	this->candidates_of_future_position.resize(8); // ���� viewpoint�� 8�� �̿� 
	predictFuturePosition();

	load_slice_set(this->slice_set, config->path_PixelRange); // ./PixelRange�� ������ �а� pixel range�� load

	LFU_Window* lfu_window;
	if (this->use_window == true)
		lfu_window = new LFU_Window(this->config, curPosX, curPosY, &state_disk_read_thread); // 3x3 ������ LFU Window, 2D list
	else
		lfu_window = new LFU_Window(this->config, curPosX, curPosY, &state_disk_read_thread, false); // 3x3 ������ LFU Window, 2D list
	this->window = lfu_window;

	state_main_thread = MAIN_THREAD_INIT; // ��׶��� Disk->Hostmem LF read�� ����ϴ� �ʱ� ������ ����
	state_h2d_thread = H2D_THREAD_INIT; // ��׶��� LRU ĳ���� ����ϴ� ������ �ʱ����
	state_disk_read_thread = DISK_READ_THREAD_NEIGHBOR_LFU_READ_COMPLETE;

	hipStreamCreate(&stream_main); // CUDA Concurrency�� ���� streams
	hipStreamCreate(&stream_h2d);

	// fill_cache();

#if PREFETCHING_MODE != 0
	workers.push_back(std::thread(&LF_Renderer::loop_nbrs_h2d, this, std::ref(*LRU), std::ref(*window), slice_set, std::ref(candidates_of_future_position), stream_h2d, std::ref(state_main_thread), std::ref(mtx)));
#endif
	// H2D, LF Read�� ���� worker threads
	if (this->use_window == true) {
		workers.push_back(std::thread(&LF_Renderer::loop_read_disk, this, std::ref(*window), std::ref(curPosX), std::ref(curPosY), std::ref(light_field_size), std::ref(state_main_thread)));
	}

	threadsPerBlock.x = 4;
	threadsPerBlock.y = 64;

	query_CudaMemory();
}

LF_Renderer::~LF_Renderer() {
	printf("Destruct LF Renderer\n");
	for (std::vector<std::thread>::iterator it = workers.begin(); it != workers.end(); it++) {
		it->join();
	}
	delete this->LRU;
	delete this->window;
#ifdef USE_UNIFIED_MEMORY
	free_uint8(synthesized_view, "unified");
#else
	free_uint8(synthesized_view, "device");
#endif
	hipStreamDestroy(stream_main);
	hipStreamDestroy(stream_h2d);
	delete this->config;
}

// ������ �Լ� 
uint8_t* LF_Renderer::do_rendering(int& newPosX, int& newPosY)
{
	if (curPosX == newPosX && curPosY == newPosY) return this->synthesized_view;
	StopWatch sw;
	sw.Start();
	if (this->use_window == false) {
		if (getLFUID(curPosX, curPosY) != getLFUID(newPosX, newPosY)) {
			printf("out-of-renderable range, return previous view\n");

			newPosX = curPosX;
			newPosY = curPosY;

			return this->synthesized_view;
		}
		else {
			this->prevprevprevPosX = prevprevPosX;
			this->prevprevprevPosY = prevprevPosY;
			this->prevprevPosX = prevPosX;
			this->prevprevPosY = prevPosY;
			this->prevPosX = curPosX;
			this->prevPosY = curPosY;
			this->curPosX = newPosX;
			this->curPosY = newPosY;
		}
	}
	else {
		this->prevprevprevPosX = prevprevPosX;
		this->prevprevprevPosY = prevprevPosY;
		this->prevprevPosX = prevPosX;
		this->prevprevPosY = prevPosY;
		this->prevPosX = curPosX;
		this->prevPosY = curPosY;
		this->curPosX = newPosX;
		this->curPosY = newPosY;

		curPosX = clamp(curPosX, 101, 499);
		curPosY = clamp(curPosY, 101, 5499);
	}

	set_rendering_params(localPosX, localPosY, output_width_each_dir, curPosX, curPosY); // CUDA ��� ������ ����, �������� ���� ���� ��

	state_main_thread = MAIN_THREAD_WAIT;
	predictFuturePosition(); // viewpoint�� �̿� 8���� ����

	while (!(getLFUID(curPosX, curPosY) == window->m_center->id && state_disk_read_thread >= DISK_READ_THREAD_CENTER_LFU_READ_COMPLETE)) {}
	printf("[%d] (%d, %d)\n", getLFUID(curPosX, curPosY), curPosX, curPosY);

	state_main_thread = MAIN_THREAD_H2D;
	mtx.lock();
	std::pair<int, int> hit_per_slice = cache_slice(*LRU, *window, slice_set, curPosX, curPosY); // ���� viewpoint�� �ʿ��� slices�� ĳ��

	int mode = LRU->synchronize_HashmapOfPtr(*window, stream_main); // Host memory <-> Device memory ����ȭ
	mtx.unlock();
	double caching_time = sw.Stop();
	printf("caching time : %f\n", caching_time);
	printf("Reuse ratio : %d / %d = %f\n", hit_per_slice.first, hit_per_slice.second, (double)hit_per_slice.first / (double)hit_per_slice.second);
	printf("Cached items : %d , FULL : %d\n", LRU->size(ODD), LRU->isFull(ODD));
	cache_validity_check(curPosX, curPosY, slice_set, window, LRU);
	sw.Start();
	state_main_thread = MAIN_THREAD_RENDERING;
	
	// output_width_each_dir --> �� ��� �̹������� ���� output width, set_rendering_range �Լ��� ȣ���ؼ� ��´�.
	// printf("output width : %d = L%d + F%d + R%d + B%d\n", output_width, output_width_each_dir[3], output_width_each_dir[0], output_width_each_dir[1], output_width_each_dir[2]);
	// launch rendering kernel
	hipError_t err;
	synthesize << < blocksPerGrid_L, threadsPerBlock, 0, stream_main >> > (synthesized_view, LRU->d_devPtr_hashmap_odd, LRU->d_devPtr_hashmap_even, 0, mode, 3, curPosX, curPosY, localPosX[3], localPosY[3], (float)config->DPP, (int)config->LF_width, (int)config->LF_height, (int)config->LF_length, (int)config->slice_width);
	synthesize << < blocksPerGrid_F, threadsPerBlock, 0, stream_main >> > (synthesized_view, LRU->d_devPtr_hashmap_odd, LRU->d_devPtr_hashmap_even, output_width_each_dir[3], mode, 0, curPosX, curPosY, localPosX[0], localPosY[0], (float)config->DPP, (int)config->LF_width, (int)config->LF_height, (int)config->LF_length, (int)config->slice_width);
	synthesize << < blocksPerGrid_R, threadsPerBlock, 0, stream_main >> > (synthesized_view, LRU->d_devPtr_hashmap_odd, LRU->d_devPtr_hashmap_even, output_width_each_dir[3] + output_width_each_dir[0], mode, 1, curPosX, curPosY, localPosX[1], localPosY[1], (float)config->DPP,(int)config->LF_width, (int)config->LF_height, (int)config->LF_length, (int)config->slice_width);
	synthesize << < blocksPerGrid_B, threadsPerBlock, 0, stream_main >> > (synthesized_view, LRU->d_devPtr_hashmap_odd, LRU->d_devPtr_hashmap_even, output_width_each_dir[3] + output_width_each_dir[0] + output_width_each_dir[1], mode, 2, curPosX, curPosY, localPosX[2], localPosY[2], (float)config->DPP, (int)config->LF_width, (int)config->LF_height, (int)config->LF_length, (int)config->slice_width);
	err = hipStreamSynchronize(stream_main);
	assert(err == hipSuccess);
	state_main_thread = MAIN_THREAD_COMPLETE;

	double rendering_time = sw.Stop();
	printf("rendering time : %f\n", rendering_time);
	
#if PREFETCHING_MODE == 0
	std::string log_path = "./experiments/prefetching/compare/ours_";
#elif PREFETCHING_MODE == 1
	std::string log_path = "./experiments/prefetching/compare/dr_";
#else 
	std::string log_path = "./experiments/prefetching/compare/none_";
#endif
	
	FILE* fh2d = fopen((log_path + std::to_string(this->config->slice_width) + ".log").c_str(), "a");
	fprintf(fh2d, "%d,%d\t%f\t%f\t%d\t%d\n", curPosX, curPosY, caching_time, rendering_time, hit_per_slice.first, hit_per_slice.second);
	fclose(fh2d);

	return synthesized_view;
}

// ���� �Լ� (worker threads join�� ����)
void LF_Renderer::terminate()
{
	state_main_thread = MAIN_THREAD_TERMINATED;
}

void LF_Renderer::load_slice_set(SliceSet slice_set[][100], std::string prefix)
{
	for (int x = 0; x < 100; x++) {
		for (int y = 0; y < 100; y++)
		{
			std::string fname = prefix + std::to_string(x) + "_" + std::to_string(y) + ".txt";
			FILE* fp = fopen(fname.c_str(), "r");

			while (!feof(fp)) {
				int dir, img, pixLn_s, pixLn_e;
				fscanf(fp, "%d\t%d\t%d\t%d\n", &dir, &img, &pixLn_s, &pixLn_e);
				SliceRange sr((FOUR_DIRECTION)dir, img, pixLn_s / config->slice_width, pixLn_e / config->slice_width);
				slice_set[x][y].push_back(sr);
			}
			fclose(fp);
		}
	}
}

void LF_Renderer::set_rendering_params(int* localPosX, int* localPosY, int* output_width, const int& curPosX, const int& curPosY)
{
	localPosX[0] = (curPosX % 100) - 50;
	localPosY[0] = (curPosY % 100) - 50;
	localPosX[1] = -1 * localPosY[0];
	localPosY[1] = localPosX[0];
	localPosX[2] = -1 * localPosX[0];
	localPosY[2] = -1 * localPosY[0];
	localPosX[3] = localPosY[0];
	localPosY[3] = -1 * localPosX[0];

	for (int i = 0; i < 4; i++) {
		float theta_L = rad2deg(atan2f((-1.0f * LFU_WIDTH / 2 - localPosX[i]), (LFU_WIDTH / 2 - localPosY[i])));
		float theta_R = rad2deg(atan2f((1.0f * LFU_WIDTH / 2 - localPosX[i]), (LFU_WIDTH / 2 - localPosY[i])));
		output_width[i] = (int)((theta_R - theta_L) / this->config->DPP);
	}

	blocksPerGrid_F.x = (int)ceil((float)output_width[0] / (float)threadsPerBlock.x);
	blocksPerGrid_R.x = (int)ceil((float)output_width[1] / (float)threadsPerBlock.x);
	blocksPerGrid_B.x = (int)ceil((float)output_width[2] / (float)threadsPerBlock.x);
	blocksPerGrid_L.x = (int)ceil((float)output_width[3] / (float)threadsPerBlock.x);
	blocksPerGrid_F.y = (int)ceil((float)(config->LF_height / 2) / (float)threadsPerBlock.y); // set a shape of the threads-per-block
	blocksPerGrid_R.y = (int)ceil((float)(config->LF_height / 2) / (float)threadsPerBlock.y); // set a shape of the threads-per-block
	blocksPerGrid_B.y = (int)ceil((float)(config->LF_height / 2) / (float)threadsPerBlock.y); // set a shape of the threads-per-block
	blocksPerGrid_L.y = (int)ceil((float)(config->LF_height / 2) / (float)threadsPerBlock.y); // set a shape of the threads-per-block
}

void LF_Renderer::predictFuturePosition()
{
#ifndef DEAD_RECKONING
	/*
	candidates_of_future_position[0] = (std::make_pair(this->curPosX, this->curPosY - this->stride));
	candidates_of_future_position[1] = (std::make_pair(this->curPosX + this->stride, this->curPosY - this->stride));
	candidates_of_future_position[2] = (std::make_pair(this->curPosX + this->stride, this->curPosY));
	candidates_of_future_position[3] = (std::make_pair(this->curPosX + this->stride, this->curPosY + this->stride));
	candidates_of_future_position[4] = (std::make_pair(this->curPosX, this->curPosY + this->stride));
	candidates_of_future_position[5] = (std::make_pair(this->curPosX - this->stride, this->curPosY + this->stride));
	candidates_of_future_position[6] = (std::make_pair(this->curPosX - this->stride, this->curPosY));
	candidates_of_future_position[7] = (std::make_pair(this->curPosX - this->stride, this->curPosY - this->stride));
	*/

	int vcX = curPosX - prevPosX;
	int vcY = curPosY - prevPosY;
	int vpX = prevPosX - prevprevPosX;
	int vpY = prevPosY - prevprevPosY;
	int vppX = prevprevPosX - prevprevprevPosX;
	int vppY = prevprevPosY - prevprevprevPosY;

	double cap = (double)this->LRU->size(ODD) / (double)this->LRU->get_slice_cache_capacity();
	candidates_of_future_position[0] = (std::make_pair(this->curPosX + vcX, this->curPosY + vcY));
	candidates_of_future_position[1] = candidates_of_future_position[0];
	candidates_of_future_position[2] = (std::make_pair(this->curPosX + vpX, this->curPosY + vpY));
	candidates_of_future_position[3] = (std::make_pair(this->curPosX + vppX, this->curPosY + vppY));
	candidates_of_future_position[4] = candidates_of_future_position[0];
	candidates_of_future_position[5] = candidates_of_future_position[0];
	candidates_of_future_position[6] = candidates_of_future_position[2];
	candidates_of_future_position[7] = candidates_of_future_position[3];
	
#else
	do_dead_reckoning(this->candidates_of_future_position, this->prevprevPosX, this->prevprevPosY, this->prevPosX, this->prevPosY, this->curPosX, this->curPosY, 120.0, 8);
	// for (int i = 0; i < candidates_of_future_position.size(); i++)
	// 	printf("\tafter %d frame : (%d, %d)\n", i + 1, candidates_of_future_position.at(i).first, candidates_of_future_position.at(i).second);
#endif
}

void LF_Renderer::loop_nbrs_h2d(LRU_Cache& LRU, const LFU_Window& window, SliceSet slice_set[][100], std::vector<std::pair<int, int>>& nbrPosition, hipStream_t stream_h2d, const MAIN_THREAD_STATE& state_main_thread, std::mutex& mtx)
{
#if PREFETCHING_MODE == 0
	while (state_main_thread != MAIN_THREAD_TERMINATED) {
		mtx.lock();
		cache_slice_in_background(LRU, window, slice_set, nbrPosition, stream_h2d, state_main_thread);
		mtx.unlock();
	}
#elif PREFETCHING_MODE == 1
	while (state_main_thread != MAIN_THREAD_TERMINATED) {
		mtx.lock();
		cache_dead_reckoning_in_background(LRU, window, slice_set, nbrPosition, stream_h2d, state_main_thread);
		mtx.unlock();
	}
#endif
}

void LF_Renderer::loop_read_disk(LFU_Window& window, const int& curPosX, const int& curPosY, const int& light_field_size, const MAIN_THREAD_STATE& state_main_thread)
{
	while (state_main_thread != MAIN_THREAD_TERMINATED) {
		int ret = window.update_window(curPosX, curPosY, light_field_size, state_main_thread);
		if (ret < 0)
			printf("Neighbor LFs read Interrupted\n");
	}
}

std::pair<int, int> LF_Renderer::cache_slice(LRU_Cache& LRU, const LFU_Window& window, SliceSet slice_set[][100], const int& posX, const int& posY)
{
	int localPosX = posX % 100;
	int localPosY = posY % 100;

	int hit_count = 0;
	int slice_count = 0;
	
	for (SliceSet::iterator it = slice_set[localPosX][localPosY].begin(); it != slice_set[localPosX][localPosY].end(); it++)
	{
		for (int slice_num = it->range_begin; slice_num <= it->range_end; slice_num++)
		{
			SliceID id;
			id.lf_number = window.m_center->LF[it->direction]->LF_number;
			id.image_number = it->image_num;
			id.slice_number = slice_num;

			size_t slice_location = find_slice_from_LF(id.image_number, id.slice_number);

			if (window.pinned_memory_status == PINNED_LFU_NOT_AVAILABLE) {
				if (window.m_center->LF[it->direction]->progress == LF_READ_PROGRESS_ODD_FIELD_PREPARED) {
					/* calculate hit_rate */
					int hit = LRU.put(id, window.m_center->LF[it->direction]->odd_field + slice_location, ODD);
					hit_count += hit;
					slice_count += 1;
					/* calculate hit_rate - end*/
				}
				else if (window.m_center->LF[it->direction]->progress == LF_READ_PROGRESS_EVEN_FIELD_PREPARED) {
					int hit = LRU.put(id, window.m_center->LF[it->direction]->odd_field + slice_location, ODD);
					LRU.put(id, window.m_center->LF[it->direction]->even_field + slice_location, EVEN);
					hit_count += hit;
					slice_count += 1;
				}
				else {
					LRU.enqueue_wait_slice(id, window.m_center->LF[it->direction]->odd_field + slice_location, ODD);
					LRU.enqueue_wait_slice(id, window.m_center->LF[it->direction]->even_field + slice_location, EVEN);
				}
			}
			else if (window.pinned_memory_status == PINNED_LFU_ODD_AVAILABLE) {
				int hit = LRU.put(id, window.m_pinnedLFU[ODD][it->direction] + slice_location, ODD);
				LRU.enqueue_wait_slice(id, window.m_pinnedLFU[EVEN][it->direction] + slice_location, EVEN);
				hit_count += hit;
				slice_count += 1;
			}
			else if (window.pinned_memory_status == PINNED_LFU_EVEN_AVAILABLE) {
				int hit = LRU.put(id, window.m_pinnedLFU[ODD][it->direction] + slice_location, ODD);
				LRU.put(id, window.m_pinnedLFU[EVEN][it->direction] + slice_location, EVEN);
				hit_count += hit;
				slice_count += 1;
			}
		}
	}

	return std::make_pair(hit_count, slice_count);
}

int LF_Renderer::cache_slice_in_background(LRU_Cache& LRU, const LFU_Window& window, SliceSet slice_set[][100], std::vector<std::pair<int, int>>& nbrPosition, hipStream_t stream_h2d, const MAIN_THREAD_STATE& thread_state_main)
{
#if 1
	int i = 0;
	int s = 0;

	while (1)
	{
		while (1) {
			for (int p = 0; p < 4; p++) {
				if (thread_state_main == MAIN_THREAD_H2D) {
					state_h2d_thread = H2D_THREAD_INTERRUPTED;
					return -1;
				} // interrupted

				int posX_at_p = nbrPosition.at(p).first % 100;
				int posY_at_p = nbrPosition.at(p).second % 100;

				if (i < slice_set[posX_at_p][posY_at_p].size())
				{
					int slice_num = slice_set[posX_at_p][posY_at_p].at(i).range_begin + s;
					int dir = slice_set[posX_at_p][posY_at_p].at(i).direction;
					int img_num = slice_set[posX_at_p][posY_at_p].at(i).image_num;

					if (slice_num <= slice_set[posX_at_p][posY_at_p].at(i).range_end) {
						SliceID id;
						id.lf_number = window.m_center->LF[dir]->LF_number;
						id.image_number = img_num;
						id.slice_number = slice_num;

						size_t slice_location = find_slice_from_LF(id.image_number, id.slice_number);
						if (window.pinned_memory_status == PINNED_LFU_ODD_AVAILABLE) {
							LRU.put(id, window.m_pinnedLFU[ODD][dir] + slice_location, stream_h2d, ODD);
						}
						if (window.pinned_memory_status == PINNED_LFU_EVEN_AVAILABLE) {
							LRU.put(id, window.m_pinnedLFU[ODD][dir] + slice_location, stream_h2d, ODD);
							LRU.put(id, window.m_pinnedLFU[EVEN][dir] + slice_location, stream_h2d, EVEN);
						}
					}
				}
			}
			i++;
			if (i >= slice_set[nbrPosition.back().first % 100][nbrPosition.back().second % 100].size()) {
				i = 0;
				break;
			}
		}
		s++;
		if (s > slice_set[nbrPosition.back().first % 100][nbrPosition.back().second % 100].back().range_end) return 0;
	}
#else
	for (int i = 0; i < nbrPosition.size(); i++)
	{
		int posX = nbrPosition.at(i).first;
		int posY = nbrPosition.at(i).second;

		for (SliceSet::iterator it = slice_set[posX % 100][posY % 100].begin(); it != slice_set[posX % 100][posY % 100].end(); it++)
		{
			if (thread_state_main == MAIN_THREAD_H2D) {
				state_h2d_thread = H2D_THREAD_INTERRUPTED;
				return -1;
			} // interrupted

			for (int slice_num = it->range_begin; slice_num <= it->range_end; slice_num++)
			{
				SliceID id;
				id.lf_number = window.m_center->LF[it->direction]->LF_number;
				id.image_number = it->image_num;
				id.slice_number = slice_num;

				size_t slice_location = find_slice_from_LF(id.image_number, id.slice_number);

				if (window.pinned_memory_status == PINNED_LFU_ODD_AVAILABLE) {
					LRU.put(id, window.m_pinnedLFU[ODD][it->direction] + slice_location, stream_h2d, ODD);
				}
				if (window.pinned_memory_status == PINNED_LFU_EVEN_AVAILABLE) {
					LRU.put(id, window.m_pinnedLFU[ODD][it->direction] + slice_location, stream_h2d, ODD);
					LRU.put(id, window.m_pinnedLFU[EVEN][it->direction] + slice_location, stream_h2d, EVEN);
				}
			}
		}
	}

	return 0;
#endif
}

size_t LF_Renderer::find_slice_from_LF(const int& img, const int& slice)
{
	return (img * config->LF_width * config->LF_height + slice * config->slice_width * config->LF_height) * 3 / 2;
}

__device__ int dev_find_pixel_location(int img, int w, int h, int width, int height, int slice_width)
{
	int slice = w / slice_width;
	int slice_number = w % slice_width;
	return img * width * height * 3 + slice * slice_width * height * 3 + slice_number * height * 3 + h * 3;
}

__device__ int dev_query_hashmap(int lf, int img, int slice, int width, int length, int slice_width)
{
	return lf * (width / slice_width) * length + img * (width / slice_width) + slice;
}

__global__ void synthesize(uint8_t* outImage, uint8_t** d_hashmap_odd, uint8_t** d_hashmap_even, int offset, int mode, int direction, int posX, int posY, int localPosX, int localPosY, float dpp, int width, int height, int legnth, int slice_width, float fov, float times)
{
	int ow = (int)(360.0f / dpp);

	int tw = blockIdx.x * blockDim.x + threadIdx.x; // blockIdx.x = (int)[0, (out_w - 1)]
	int th = blockIdx.y * blockDim.y + threadIdx.y; // threadIdx = (int)[0, (g_height - 1)]

	int LFUW = 100;
	int DATAW = 50;
	int Y = LFUW / 2;

	float theta_L = dev_rad2deg(atan2f((-1.0f * LFUW / 2 - localPosX), (LFUW / 2 - localPosY)));
	float theta_R = dev_rad2deg(atan2f((1.0f * LFUW / 2 - localPosX), (LFUW / 2 - localPosY)));
	if (localPosY == 50) {
		theta_L = -90.0f;
		theta_R = 90.0f;
	}

	int output_width = (int)((theta_R - theta_L) / dpp);

	if (tw < output_width && th < (height >> 1)) // Thread index must not exceed output resolution
	{
		float theta_P = theta_L + (dpp * (float)tw);

		float b = sqrt(2.0f) * LFUW;
		float xP = (float)(Y - localPosY) * tanf(dev_deg2rad(theta_P)) + localPosX;
		float N_dist = sqrt((float)((xP - localPosX) * (xP - localPosX) + (Y - localPosY) * (Y - localPosY))) / b;

		xP /= 2;
		int P_1 = (int)(roundf(xP + (DATAW >> 1)));
		if (direction == 1 || direction == 2) {
			P_1 = DATAW - P_1 - 1;
		}
		P_1 = dev_Clamp(P_1, 0, DATAW - 1);

		float U = (theta_P * (1.0f / 180.0f)) * (width >> 1) + (width >> 1);
		int U_1 = (int)(roundf(U));
		if (direction == 1) U_1 += width >> 2;
		if (direction == 2) U_1 += width >> 1;
		if (direction == 3) U_1 -= width >> 2;
		if (U_1 >= width) U_1 = U_1 - width;
		else if (U_1 < 0) U_1 = U_1 + width;
		U_1 = dev_Clamp(U_1, 0, width - 1);

		int N_off = (int)(roundf(times * N_dist + 0.5)) >> 1;

		int LF_num = dev_find_LF_number_BMW(direction, posX, posY);
		int image_num = P_1 % legnth;
		int slice_num = U_1 / slice_width;
		int pixel_col = U_1 % slice_width;

		float N_H_r = (float)(height + N_off) / height;

		float h_n = (th - height / 2) * N_H_r + height / 2;

		if (h_n < 0)
			h_n = (-1 * h_n) - 1;
		else if (h_n > height - 1)
			h_n = height - ((h_n - height) - 1);

		int H_1 = (int)(roundf(h_n));
		H_1 = dev_Clamp(H_1, 0, height - 1);
		float H_r = h_n - H_1;

		int slice = dev_query_hashmap(LF_num, image_num, slice_num, width, legnth, slice_width); // Random access to hashmap

		uint8_t oddpel_ch0 = d_hashmap_odd[slice][(pixel_col * (height >> 1)) * 3 + H_1 * 3 + 2]; // Random access to pixel column
		uint8_t oddpel_ch1 = d_hashmap_odd[slice][(pixel_col * (height >> 1)) * 3 + H_1 * 3 + 1]; // Random access to pixel column
		uint8_t oddpel_ch2 = d_hashmap_odd[slice][(pixel_col * (height >> 1)) * 3 + H_1 * 3 + 0]; // Random access to pixel column
		outImage[((2 * th) * (ow * 3) + offset * 3) + tw * 3 + 0] = oddpel_ch0; // b 
		outImage[((2 * th) * (ow * 3) + offset * 3) + tw * 3 + 1] = oddpel_ch1; // g 
		outImage[((2 * th) * (ow * 3) + offset * 3) + tw * 3 + 2] = oddpel_ch2; // r 

		if (mode == 1) {
			uint8_t evenpel_ch0 = d_hashmap_even[slice][(pixel_col * (height >> 1)) * 3 + H_1 * 3 + 2]; // Random access to pixel column
			uint8_t evenpel_ch1 = d_hashmap_even[slice][(pixel_col * (height >> 1)) * 3 + H_1 * 3 + 1]; // Random access to pixel column
			uint8_t evenpel_ch2 = d_hashmap_even[slice][(pixel_col * (height >> 1)) * 3 + H_1 * 3 + 0]; // Random access to pixel column

			outImage[((2 * th + 1) * (ow * 3) + offset * 3) + tw * 3 + 0] = evenpel_ch0; // b 
			outImage[((2 * th + 1) * (ow * 3) + offset * 3) + tw * 3 + 1] = evenpel_ch1; // g 
			outImage[((2 * th + 1) * (ow * 3) + offset * 3) + tw * 3 + 2] = evenpel_ch2; // r
		}
		else
		{
			outImage[((2 * th + 1) * (ow * 3) + offset * 3) + tw * 3 + 0] = oddpel_ch0; // b 
			outImage[((2 * th + 1) * (ow * 3) + offset * 3) + tw * 3 + 1] = oddpel_ch1; // g 
			outImage[((2 * th + 1) * (ow * 3) + offset * 3) + tw * 3 + 2] = oddpel_ch2; // r 
		}
#if 0
		if (tw == 0) // for debug
		{
			outImage[((2 * th) * (ow * 3) + offset * 3) + tw * 3 + 0] = 0; // b 
			outImage[((2 * th) * (ow * 3) + offset * 3) + tw * 3 + 1] = 0; // g 
			outImage[((2 * th) * (ow * 3) + offset * 3) + tw * 3 + 2] = 255; // r 
			outImage[((2 * th + 1) * (ow * 3) + offset * 3) + tw * 3 + 0] = 0; // b 
			outImage[((2 * th + 1) * (ow * 3) + offset * 3) + tw * 3 + 1] = 0; // g 
			outImage[((2 * th + 1) * (ow * 3) + offset * 3) + tw * 3 + 2] = 255; // r
		}
		if (tw == output_width - 1) // for debug
		{
			outImage[((2 * th) * (ow * 3) + offset * 3) + tw * 3 + 0] = 255; // b 
			outImage[((2 * th) * (ow * 3) + offset * 3) + tw * 3 + 1] = 0; // g 
			outImage[((2 * th + 1) * (ow * 3) + offset * 3) + tw * 3 + 0] = 255; // b 
			outImage[((2 * th + 1) * (ow * 3) + offset * 3) + tw * 3 + 1] = 0; // g 
		}
#endif
	}
}

// function for debugging
void cache_validity_check(int curPosX, int curPosY, SliceSet slice_set[][100], LFU_Window* window, LRU_Cache* LRU)
{
	bool error = false;
	int required_slices = slice_set[curPosX % 100][curPosY % 100].size();
	printf("%d slices are required\n", required_slices);
	for (SliceSet::iterator ss = slice_set[curPosX % 100][curPosY % 100].begin(); ss != slice_set[curPosX % 100][curPosY % 100].end(); ss++)
	{
		for (int sn = ss->range_begin; sn <= ss->range_end; sn++)
		{
			SliceID test_id;
			test_id.lf_number = window->m_center->LF[ss->direction]->LF_number;
			test_id.image_number = ss->image_num;
			test_id.slice_number = sn;
			
// 			int hasmap_idx = test_id.lf_number * (LF_width / config->slice_width) * config->LF_length + test_id.image_number * (config->LF_width / config->slice_width) + test_id.slice_number;

			uint8_t* test_addr = LRU->find_slice_in_hashmap(test_id);
			if (test_addr == nullptr) {
				printf("[%d-%d-%d] is not exist\n", test_id.lf_number, test_id.image_number, test_id.slice_number);
				error = true;
			}
			else {
				// printf("[%d-%d-%d] test_addr : %x\n", test_id.lf_number, test_id.image_number, test_id.slice_number, test_addr);
				// uint8_t* test_val = LRU->hashmap_odd[LRU->query_hashmap(test_id, ODD)]->odd_data; // back data���� � ��?
			}
		}
	}
	if (error) exit(1);
}

void LF_Renderer::generate_LFConfig(const std::string& path_LF, const std::string& path_pixelrange, const size_t& iw, const size_t& ih, const size_t& lf_len, const size_t& numLFs, const double& dpp)
{
	size_t dm = (size_t)((double)get_devmem_freespace() * 0.8);
	size_t max_pixel_range = (size_t)ceil(2.0 * rad2deg(atan2(0.5, 1.0)) * iw / 360.0);
	size_t neareast_mpr;
	size_t sw_upper_bound = 0;
	double mindiff = 1e6;

	for (int n = 1; n <= iw; n++) {
		if (iw % n == 0) {
			size_t sw = iw / n; // assume a slice width

			if (abs(max_pixel_range - (double)sw) < mindiff)
			{
				mindiff = abs(max_pixel_range - (double)sw);
				neareast_mpr = sw;
			}

			size_t dm_hashmap = (iw / sw) * lf_len * numLFs * sizeof(uint8_t*);
			size_t dm_slice = dm - dm_hashmap - (size_t)(360.0 /dpp) * ih * 3;
			size_t cache_capacity = dm_slice / (sw * ih * 3);
			if (cache_capacity > (4 * lf_len) * 1.2)
				sw_upper_bound = sw_upper_bound < sw ? sw : sw_upper_bound;
		}
	}
	size_t optiman_sw = std::min(neareast_mpr, sw_upper_bound);

	this->config = new LF_Config(path_LF, path_pixelrange, iw, ih, lf_len, numLFs, optiman_sw, dpp);
}

std::pair<double, double> LF_Renderer::deadReckoning(std::pair<double, double> a_2, std::pair<double, double> a_1, std::pair<double, double> a0, double framerate, int f)
{
	double tf = (double)f / framerate;

	std::pair<double, double> v0;
	std::pair<double, double> v_1;
	std::pair<double, double> c0;

	v0 = std::make_pair(differentiation(a_1.first, a0.first, 1.0 / framerate), differentiation(a_1.second, a0.second, 1.0 / framerate));
	v_1 = std::make_pair(differentiation(a_2.first, a_1.first, 1.0 / framerate), differentiation(a_2.second, a_1.second, 1.0 / framerate));
	c0 = std::make_pair(differentiation(v_1.first, v0.first, tf), differentiation(v_1.second, v0.second, tf));

	return std::make_pair(a0.first + v0.first * tf + 0.5 * c0.first * tf * tf, a0.second + v0.second * tf + 0.5 * c0.second * tf * tf);;
}

void LF_Renderer::do_dead_reckoning(std::vector<std::pair<int, int>>& candidates, double prevprevPosX, double prevprevPosY, double prevPosX, double prevPosY, double curPosX, double curPosY, double framerate, int prediction_range)
{
	std::pair<double, double> prevprevPos = std::make_pair(prevprevPosX, prevprevPosY);
	std::pair<double, double> prevPos = std::make_pair(prevPosX, prevPosY);
	std::pair<double, double> curPos = std::make_pair(curPosX, curPosY);

	for (int i = 1; i <= prediction_range; i++)
	{
		std::pair<double, double> nextPos = deadReckoning(prevprevPos, prevPos, curPos, framerate, i);
		nextPos.first = (double)clamp(nextPos.first, 1, ((int)curPosX / 100 + 1) * 100 - 1);
		nextPos.second = (double)clamp(nextPos.second, 1, ((int)curPosY / 100 + 1) * 100 - 1);
		candidates.at(i-1) = (std::make_pair((int)nextPos.first, (int)nextPos.second));
	}
}

int LF_Renderer::cache_dead_reckoning_in_background(LRU_Cache& LRU, const LFU_Window& window, SliceSet slice_set[][100], std::vector<std::pair<int, int>>& nbrPosition, hipStream_t stream_h2d, const MAIN_THREAD_STATE& thread_state_main)
{
	for (int i = 0; i < nbrPosition.size(); i++)
	{
		int posX = nbrPosition.at(i).first;
		int posY = nbrPosition.at(i).second;

		for (SliceSet::iterator it = slice_set[posX % 100][posY % 100].begin(); it != slice_set[posX % 100][posY % 100].end(); it++)
		{
			if (thread_state_main == MAIN_THREAD_H2D) {
				state_h2d_thread = H2D_THREAD_INTERRUPTED;
				return -1;
			} // interrupted

			for (int slice_num = it->range_begin; slice_num <= it->range_end; slice_num++)
			{
				SliceID id;
				id.lf_number = window.m_center->LF[it->direction]->LF_number;
				id.image_number = it->image_num;
				id.slice_number = slice_num;

				size_t slice_location = find_slice_from_LF(id.image_number, id.slice_number);

				if (window.pinned_memory_status == PINNED_LFU_ODD_AVAILABLE) {
					LRU.put(id, window.m_pinnedLFU[ODD][it->direction] + slice_location, stream_h2d, ODD);
				}
				if (window.pinned_memory_status == PINNED_LFU_EVEN_AVAILABLE) {
					LRU.put(id, window.m_pinnedLFU[ODD][it->direction] + slice_location, stream_h2d, ODD);
					LRU.put(id, window.m_pinnedLFU[EVEN][it->direction] + slice_location, stream_h2d, EVEN);
				}
			}
		}
	}

	return 0;
}

void LF_Renderer::fill_cache()
{
	for (int i = 0; i < 100; i++) {
		for (int j = 0; j < 100; j++) {
			cache_slice(*LRU, *window, slice_set, i, j); 
			if (LRU->isFull(ODD) == true) break;
		}
	}
}