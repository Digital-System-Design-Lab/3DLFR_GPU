#include "Device_Memory_Manager.cuh"

DeviceMemoryManager::DeviceMemoryManager(const size_t& num_of_slice, const size_t& slice_size)
{
	hipError_t err;
	err = hipMalloc((void**)&dev_slice_buffer, slice_size * num_of_slice);
	assert(err == hipSuccess);
	access_number_set.resize(num_of_slice);
	this->num_of_slice = num_of_slice;
	this->slice_size = slice_size;
	this->last_access_number = 0;
}

DeviceMemoryManager::~DeviceMemoryManager()
{
	hipError_t err;
	err = hipFree(dev_slice_buffer);
	assert(err == hipSuccess);
}

int DeviceMemoryManager::rent_access_number()
{
	for (int i = last_access_number; i < num_of_slice; i++) {
		if (access_number_set[i] < 1) {
			access_number_set[i] = 1;
			last_access_number = i;

			return i;
		}
	}
}
void DeviceMemoryManager::return_access_number(const size_t& num)
{
	access_number_set[num] = 0;
}

uint8_t* DeviceMemoryManager::get_empty_space(const size_t& access_number)
{
	return dev_slice_buffer + access_number * slice_size;
}

void DeviceMemoryManager::init()
{
	hipMemset(this->dev_slice_buffer, 0, slice_size * num_of_slice);
	this->last_access_number = 0;
}