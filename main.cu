#include "hip/hip_runtime.h"
﻿#include "LFUtils.cuh"
#include <thread> // std::thread
#include <future> // std::future
#include <mutex>
#include <fcntl.h> // file open flag
#include <io.h> // file descriptor
#include <assert.h> // assert
#include <stdlib.h> // size_t
#include <stdint.h> // uint8_t
#include <map>
#include <list>

#define LOGGER 1

std::vector<std::vector<std::pair<int, int>>> slice_map(50);

__device__ int dev_find_pixel_location(int img, int w, int h, int g_width, int g_height, int g_slice_width)
{
	int slice = w / g_slice_width;
	int slice_number = w % g_slice_width;
	return img * g_width * g_height * 3 + slice * g_slice_width * g_height * 3 + slice_number * g_height * 3 + h * 3;
}

__device__ int dev_query_hashmap(const int& lf, const int& img, const int& slice)
{
	return lf * (g_width / g_slice_width) * g_length + img * (g_width / g_slice_width) + slice;
}

__global__ void rendering(uint8_t* outImage, uint8_t** d_hashmap_odd, uint8_t** d_hashmap_even, int mode, int posX, int posY, int g_width, int g_height, int g_slice_width, float fov = 90.0f, float times = 270.0f)
{
	int tw = blockIdx.x * blockDim.x + threadIdx.x; // blockIdx.x = (int)[0, (out_w - 1)]
	int th = blockIdx.y * blockDim.y + threadIdx.y; // threadIdx = (int)[0, (g_height - 1)]

	int LFUW = 100;
	int z0 = posY; 
	int x0 = posX; 

	float theta_L = -fov / 2.0;
	float theta_R = fov / 2.0;

	float theta_P = theta_L + (0.04 * (float)tw);
	int Y = LFUW / 2;
	float b = sqrt(2.0) * LFUW;
	float xP = x0 + z0 * __tanf(dev_deg2rad(theta_P)); 

	float N_dist = sqrt((float)((xP - x0) * (xP - x0) + (Y - z0) * (Y - z0))) / b; 
	int P_1 = (int)(roundf(xP));
	float U = (theta_P / (fov / 2.0)) * WIDTH / 2 + WIDTH / 2;

	int U_1 = (int)(roundf(U));
	int U_1_n = 0;
	int N_off = (int)(roundf(times * N_dist + 0.5)) >> 1;
	
	U_1 %= WIDTH;
	U_1 = dev_Clamp(U_1, 0, WIDTH - 1);

	int LF_num = P_1 / LENGTH;
	int image_num = P_1 % LENGTH;
	int slice_num = U_1 / g_slice_width;
	int pixel_col = U_1 % g_slice_width;
	
	float N_H_r = (float)(HEIGHT + N_off) / HEIGHT;
	
	float h_n = (th - HEIGHT / 2) * N_H_r + HEIGHT / 2;

	if (h_n < 0)
		h_n = (-1 * h_n) - 1;
	else if (h_n > HEIGHT - 1)
		h_n = HEIGHT - ((h_n - HEIGHT) - 1);

	int H_1 = (int)(roundf(h_n));
	H_1 = dev_Clamp(H_1, 0, HEIGHT - 1);
	float H_r = h_n - H_1;

	int slice = dev_query_hashmap(LF_num, image_num, slice_num); // Random access to hashmap
	uint8_t oddpel_ch0 = d_hashmap_odd[slice][(pixel_col * g_height / 2) * 3 + H_1 * 3 + 0]; // Random access to pixel column
	uint8_t oddpel_ch1 = d_hashmap_odd[slice][(pixel_col * g_height / 2) * 3 + H_1 * 3 + 1]; // Random access to pixel column
	uint8_t oddpel_ch2 = d_hashmap_odd[slice][(pixel_col * g_height / 2) * 3 + H_1 * 3 + 2]; // Random access to pixel column
	outImage[(2 * th) * (OUTPUT_WIDTH * 3) + tw * 3 + 0] = oddpel_ch0; // b 
	outImage[(2 * th) * (OUTPUT_WIDTH * 3) + tw * 3 + 1] = oddpel_ch1; // g 
	outImage[(2 * th) * (OUTPUT_WIDTH * 3) + tw * 3 + 2] = oddpel_ch2; // r 

	if (mode == 1) {
		uint8_t evenpel_ch0 = d_hashmap_even[slice][(pixel_col * g_height / 2) * 3 + H_1 * 3 + 0]; // Random access to pixel column
		uint8_t evenpel_ch1 = d_hashmap_even[slice][(pixel_col * g_height / 2) * 3 + H_1 * 3 + 1]; // Random access to pixel column
		uint8_t evenpel_ch2 = d_hashmap_even[slice][(pixel_col * g_height / 2) * 3 + H_1 * 3 + 2]; // Random access to pixel column

		outImage[(2 * th + 1) * (OUTPUT_WIDTH * 3) + tw * 3 + 0] = evenpel_ch0; // b 
		outImage[(2 * th + 1) * (OUTPUT_WIDTH * 3) + tw * 3 + 1] = evenpel_ch1; // g 
		outImage[(2 * th + 1) * (OUTPUT_WIDTH * 3) + tw * 3 + 2] = evenpel_ch2; // r 
	}
	else
	{
		outImage[(2 * th + 1) * (OUTPUT_WIDTH * 3) + tw * 3 + 0] = oddpel_ch0; // b 
		outImage[(2 * th + 1) * (OUTPUT_WIDTH * 3) + tw * 3 + 1] = oddpel_ch1; // g 
		outImage[(2 * th + 1) * (OUTPUT_WIDTH * 3) + tw * 3 + 2] = oddpel_ch2; // r 
	}

}

uint8_t* alloc_uint8(int size, std::string alloc_type) {
	uint8_t* buf;
	if (alloc_type == "pinned") {
		hipHostMalloc((void**)&buf, size);
		memset(buf, 0, size);
	}
	else if (alloc_type == "pageable") {
		buf = new uint8_t[size]();
		memset(buf, 0, size);

	}
	else if (alloc_type == "device") {
		hipMalloc((void**)&buf, size);
		hipMemset(buf, 0, size);
	}
	else if (alloc_type == "unified") {
		hipMallocManaged((void**)&buf, size);
	}
	else exit(1);

	return buf;
}

void free_uint8(uint8_t* buf, std::string alloc_type) {
	if (alloc_type == "pinned") {
		hipHostFree(buf);
	}
	else if (alloc_type == "pageable") {
		delete[] buf;
	}
	else if (alloc_type == "device" || alloc_type == "unified") {
		hipFree(buf);
	}
	else exit(1);
}

int read_uint8(uint8_t* buf, std::string filename, int size = -1)
{
	int fd;
	int ret;

	fd = open(filename.c_str(), O_RDONLY | O_BINARY);
	ret = fd;
	if (ret < 0) {
		printf("open failed, %s\n", filename.c_str());
		assert(ret == 0);
		exit(1);
	}

	if (size < 0) {
		if ((ret = lseek(fd, 0, SEEK_END)) < 0) {
			printf("SEEK_END failed, %s\n", filename.c_str());
			assert(ret == 0);
			exit(1);
		}
		if ((ret = tell(fd)) < 0) {
			printf("tell failed, %s\n", filename.c_str());
			assert(ret == 0);
			exit(1);
		}
		size = ret;
		if ((ret = lseek(fd, 0, SEEK_SET)) < 0) {
			printf("SEEK_SET failed, %s\n", filename.c_str());
			assert(ret == 0);
			exit(1);
		}
	}

	ret = read(fd, buf, sizeof(uint8_t) * size); // x64
	close(fd);

	if (ret != size) {
		printf("read failed, %s\n", filename.c_str());
		assert(ret == size);
		exit(1);
	}

	return ret;
}

int write_uint8(uint8_t* buf, std::string filename, int size = -1)
{
	int fd;
	if ((fd = open(filename.c_str(), O_WRONLY | O_BINARY)) < 0) return fd;
	if (size < 0) size = _msize(buf);

	int ret = write(fd, buf, sizeof(uint8_t) * size); // x64 
	close(fd);

	return ret;
}

void set_slice_map() {
	for (int y = 1; y <= 49; y++)
	{
		std::string sidLogFile = "S:/len50/" + std::to_string(5) + "K/log2/" + std::to_string(50) + "_" + std::to_string(y) + ".txt";
		FILE* sidLog = fopen(sidLogFile.c_str(), "r");

		while (!feof(sidLog)) {
			int img, pixLn_s, pixLn_e;
			fscanf(sidLog, "%d\t%d\t%d\n", &img, &pixLn_s, &pixLn_e);
			slice_map[y].push_back(std::make_pair(pixLn_s / g_slice_width, pixLn_e / g_slice_width));
		}
		fclose(sidLog);
	}
}

void getNeighborList(std::vector<std::pair<int, int>>& nbrPosition, int curPosX, int curPosY)
{
	nbrPosition.at(0) = (std::make_pair(curPosX, curPosY - 1));
	nbrPosition.at(1) = (std::make_pair(curPosX + 1, curPosY - 1));
	nbrPosition.at(2) = (std::make_pair(curPosX + 1, curPosY));
	nbrPosition.at(3) = (std::make_pair(curPosX + 1, curPosY + 1));
	nbrPosition.at(4) = (std::make_pair(curPosX, curPosY + 1));
	nbrPosition.at(5) = (std::make_pair(curPosX - 1, curPosY + 1));
	nbrPosition.at(6) = (std::make_pair(curPosX - 1, curPosY));
	nbrPosition.at(7) = (std::make_pair(curPosX - 1, curPosY - 1));
}

void set_both_end_image(int& leftend_image, int& rightend_image, const int& posX, const int& posY)
{
	leftend_image = (posX - posY);
	rightend_image = (posX + posY);
	if (leftend_image < 0 || posY <= 0 || posX <= 0) {
		printf("OUT OF RENDERABLE AREA\n");
		exit(1);
	}
}

std::pair<size_t, size_t> cache_slice(LRUCache& LRU, std::vector<Interlaced_LF>& window, const int& posX, const int& posY) {

	size_t hit = 0;
	size_t try_caching = 0;

	int leftend_image, rightend_image;
	set_both_end_image(leftend_image, rightend_image, posX, posY);

	int img = leftend_image;

	for (std::vector<std::pair<int, int>>::iterator image_iter = slice_map[posY].begin(); image_iter != slice_map[posY].end(); image_iter++) 
	{
		SliceID id;
		for (int slice_num = image_iter->first; slice_num <= image_iter->second; slice_num++) {
			id.lf_number = img / g_length;
			id.image_number = img % g_length;
			id.slice_number = slice_num;

			int slice_location = find_slice_from_LF(id.image_number, id.slice_number, true);
			// uint8_t* data;
			Interlaced_LF* LF = get_LF_from_Window(window, id.lf_number);

			if (LF->progress < LF_READ_PROGRESS_ODD_FIELD_PREPARED) { 
				LRU.enqueue_wait_slice(id, LF->odd_field + slice_location, ODD);
				LRU.enqueue_wait_slice(id, LF->even_field + slice_location, EVEN);
			}
			else if (LF->progress == LF_READ_PROGRESS_ODD_FIELD_PREPARED) { 
				LRU.put(id, LF->odd_field + slice_location, ODD);
				LRU.enqueue_wait_slice(id, LF->even_field + slice_location, EVEN);
			}
			else { 
				LRU.put(id, LF->odd_field + slice_location, ODD);
				LRU.put(id, LF->even_field + slice_location, EVEN);
			}

			try_caching++;
		}
		img++;
	}
	return std::make_pair(hit, try_caching);
}

int cache_slice_in_background(LRUCache& LRU, std::vector<Interlaced_LF>& window, std::vector<std::pair<int, int>>& nbrPosition, hipStream_t stream_h2d, H2D_THREAD_STATE& thread_state_h2d, const MAIN_THREAD_STATE& thread_state_main) {

	int i = 0;
	int s = 0;
	while (1)
	{
		while (1) {
			for (int p = 0; p < 8; p++) {
				if (thread_state_main < MAIN_THREAD_RENDERING) {
					thread_state_h2d = H2D_THREAD_INTERRUPTED;
					return -1;
				} // interrupted

				int posX_at_p = nbrPosition.at(p).first;
				int posY_at_p = nbrPosition.at(p).second;

				int leftend_image, rightend_image;
				set_both_end_image(leftend_image, rightend_image, posX_at_p, posY_at_p);

				int img = leftend_image + i;
				std::pair<int, int> slice_range = slice_map[posY_at_p].at(i); 
				int slice_num = slice_range.first + s; 

				if (i < slice_map[posY_at_p].size() && slice_num <= slice_range.second) 
				{
					SliceID id;

					id.lf_number = img / g_length;
					id.image_number = img % g_length;
					id.slice_number = slice_num;

					int slice_location = find_slice_from_LF(id.image_number, id.slice_number, true);
					Interlaced_LF* LF = get_LF_from_Window(window, id.lf_number);

					if (LF->progress == LF_READ_PROGRESS_ODD_FIELD_PREPARED) {
						LRU.put(id, LF->odd_field + slice_location, stream_h2d, thread_state_h2d, ODD);
					}
					if (LF->progress == LF_READ_PROGRESS_EVEN_FIELD_PREPARED) {
						LRU.put(id, LF->odd_field + slice_location, stream_h2d, thread_state_h2d, ODD);
						LRU.put(id, LF->even_field + slice_location, stream_h2d, thread_state_h2d, EVEN);
					}
				}
			}
			i++;
			if (i >= slice_map[nbrPosition.back().second].size()) {
				i = 0;
				break;
			}
		}
		s++;
		if (s > slice_map[nbrPosition.back().second].back().second) return 0;
	}
}

void loop_nbrs_h2d(LRUCache& LRU, std::vector<Interlaced_LF>& window, std::vector<std::pair<int, int>>& nbrPosition, hipStream_t stream_h2d, H2D_THREAD_STATE& thread_state_h2d, const MAIN_THREAD_STATE& thread_state_main, std::mutex& mtx)
{
	bool loop = true;
	while (loop) {
		mtx.lock();
		cache_slice_in_background(LRU, window, nbrPosition, stream_h2d, thread_state_h2d, thread_state_main);
		mtx.unlock();
		if (thread_state_main == MAIN_THREAD_TERMINATED) loop = false;
	}
}

void update_LF_window(std::vector<Interlaced_LF>& window, int& current_LF_number, const int& curPosX, READ_DISK_THREAD_STATE& read_disk_thread_state)
{
	StopWatch sw_read;
	int assumed_read_time_for_field = 3000;

	std::string prefix = g_directory + "Interlaced/Column";

	current_LF_number = curPosX / g_length;
	Interlaced_LF* curLF = get_LF_from_Window(window, current_LF_number);

	if (curLF->progress < LF_READ_PROGRESS_EVEN_FIELD_PREPARED) {
		read_disk_thread_state = READ_DISK_THREAD_CURRENT_LF_READING;
		printf("Current LF is not read yet\n");
		if (curLF->progress < LF_READ_PROGRESS_ODD_FIELD_PREPARED) {
			sw_read.Start();
			read_uint8(curLF->odd_field, prefix + std::to_string(current_LF_number) + "_odd.bgr");
			_sleep(assumed_read_time_for_field - sw_read.Stop());
			curLF->progress = LF_READ_PROGRESS_ODD_FIELD_PREPARED;
		}
		else {
			sw_read.Start();
			read_uint8(curLF->even_field, prefix + std::to_string(current_LF_number) + "_even.bgr");
			_sleep(assumed_read_time_for_field - sw_read.Stop());
			curLF->progress = LF_READ_PROGRESS_EVEN_FIELD_PREPARED;
		}
		read_disk_thread_state = READ_DISK_THREAD_CURRENT_LF_READ_COMPLETE;
	}

	int leftend_LF = current_LF_number - 1 < 0 ? 0 : current_LF_number - 1;
	int rightend_LF = leftend_LF + g_LF_window_size - 1;

	if (leftend_LF > window.front().LF_number) {
		// LF Window has been slided to right
		read_disk_thread_state = READ_DISK_THREAD_NEIGHBOR_LF_READING;
		printf("move right, start LF reading in the background\n");

		Interlaced_LF tmp = window.front();
		for (std::vector<Interlaced_LF>::iterator iter = window.begin(); iter != window.end() - 1; iter++) {
			*iter = *(iter + 1);
		}
		window.back() = tmp;
		window.back().LF_number = rightend_LF;
		window.back().progress = LF_READ_PROGRESS_NOT_PREPARED;
		// read_uint8(window.back().full_field, prefix + std::to_string(rightend_LF) + ".bgr");

		sw_read.Start();
		read_uint8(window.back().odd_field, prefix + std::to_string(rightend_LF) + "_odd.bgr");
		_sleep(assumed_read_time_for_field - sw_read.Stop());
		window.back().progress = LF_READ_PROGRESS_ODD_FIELD_PREPARED;

		sw_read.Start();
		read_uint8(window.back().even_field, prefix + std::to_string(rightend_LF) + "_even.bgr");
		_sleep(assumed_read_time_for_field - sw_read.Stop());
		window.back().progress = LF_READ_PROGRESS_EVEN_FIELD_PREPARED;

		read_disk_thread_state = READ_DISK_THREAD_NEIGHBOR_LF_READ_COMPLETE;

		printf("read in background complete (right), Assume that reading needs %d ms\n", assumed_read_time_for_field * 2);
	}
	else if (leftend_LF < window.front().LF_number) {
		// LF Window has been slided to left
		read_disk_thread_state = READ_DISK_THREAD_NEIGHBOR_LF_READING;
		printf("move left, read in background\n");
		sw_read.Start();
		Interlaced_LF tmp = window.back();
		for (std::vector<Interlaced_LF>::iterator iter = window.end() - 1; iter != window.begin(); iter--) {
			*iter = *(iter - 1);
		}
		window.front() = tmp;
		window.front().LF_number = leftend_LF;
		window.front().progress = LF_READ_PROGRESS_NOT_PREPARED;

		// read_uint8(window.front().full_field, prefix + std::to_string(leftend_LF) + ".bgr");
		sw_read.Start();
		read_uint8(window.front().odd_field, prefix + std::to_string(leftend_LF) + "_odd.bgr");
		_sleep(assumed_read_time_for_field - sw_read.Stop());
		window.front().progress = LF_READ_PROGRESS_ODD_FIELD_PREPARED;

		sw_read.Start();
		read_uint8(window.front().even_field, prefix + std::to_string(leftend_LF) + "_even.bgr");
		_sleep(assumed_read_time_for_field - sw_read.Stop());
		window.front().progress = LF_READ_PROGRESS_EVEN_FIELD_PREPARED;

		read_disk_thread_state = READ_DISK_THREAD_NEIGHBOR_LF_READ_COMPLETE;

		printf("read in background complete (left), Assume that reading needs %d ms\n", assumed_read_time_for_field * 2);
	}
}

void loop_read_disk(std::vector<Interlaced_LF>& window, int& current_center_of_LF_window, const int& curPosX, READ_DISK_THREAD_STATE& read_disk_thread_state, const MAIN_THREAD_STATE& main_thread_state)
{
	bool loop = true;
	while (loop) {
		update_LF_window(window, current_center_of_LF_window, curPosX, read_disk_thread_state);
		if (main_thread_state == MAIN_THREAD_TERMINATED) loop = false;
	}
}

int main()
{
	/* Declare */
	StopWatch sw; // for benchmark
	const int limit_cached_slice = 500;
	const int limit_hashing_LF = 50;

	printf("Input resolution : %dx%dx%d\n", g_width, g_height, g_length);
	printf("Output resolution : %dx%d\n", g_output_width, g_height);
	printf("Slice resolution : %dx%d\n", g_slice_width, g_height);
	printf("Slice Cache Size Limit : %f MB\n", g_slice_size * limit_cached_slice / 1e6);
	printf("Hashing LF Range Limit : %d to %d\n", 0, limit_hashing_LF);

	LRUCache LRU(limit_hashing_LF, limit_cached_slice);

	hipStream_t stream_main, stream_h2d;
	hipStreamCreate(&stream_main);
	hipStreamCreate(&stream_h2d);

	const int light_field_size = g_width * g_height *g_length * 3;
	uint8_t* u_synthesized_view = alloc_uint8(g_output_width * g_height * 3, "unified");

	std::vector<std::vector<Slice>> required_slices_at_eight_nbrs(8);

	int current_LF_number;
	int curPosX, curPosY;
	int prvPosX, prvPosY;
	std::vector<std::pair<int, int>> nbrPosition(8);

	std::vector<Interlaced_LF> LF_window(g_LF_window_size);

	for (int i = 0; i < g_LF_window_size; i++) {
		// LF_window.at(i).full_field = alloc_uint8(light_field_size, "pinned");
		LF_window.at(i).odd_field = alloc_uint8(light_field_size / 2, "pinned");
		LF_window.at(i).even_field = alloc_uint8(light_field_size / 2, "pinned");
	}

	/* Initialize */
	curPosX = 101;
	curPosY = 24;
	int prevPosX = curPosX;
	int prevPosY = curPosY;
	int prevprevPosX = curPosX;
	int prevprevPosY = curPosY;

	current_LF_number = curPosX / g_length; // readdisk thread updated
	int leftend_LF = current_LF_number - 1 < 0 ? 0 : current_LF_number - 1;
	int rightend_LF = leftend_LF + g_LF_window_size - 1;

	for (int i = 0; i < g_LF_window_size; i++)
	{
		LF_window.at(i).LF_number = leftend_LF + i;
		// read_uint8(LF_window.at(i).full_field, (g_directory + "Full/Column" + std::to_string(leftend_LF + i) + ".bgr"));
		read_uint8(LF_window.at(i).odd_field, (g_directory + "Interlaced/Column" + std::to_string(leftend_LF + i) + "_odd.bgr"));
		read_uint8(LF_window.at(i).even_field, (g_directory + "Interlaced/Column" + std::to_string(leftend_LF + i) + "_even.bgr"));
		LF_window.at(i).progress = LF_READ_PROGRESS_EVEN_FIELD_PREPARED;
	}

	set_slice_map(); 

	int twid = 2;
	int thei = 32;
	dim3 threadsPerBlock(twid, thei); 
	// interlace mode -> block shape : 2250*1280
	dim3 blocksPerGrid((int)ceil((float)g_output_width / (float)twid), (int)ceil((float)(g_height / 2) / (float)thei)); // set a shape of the threads-per-block


	MAIN_THREAD_STATE state_main_thread;
	H2D_THREAD_STATE state_h2d_thread;
	READ_DISK_THREAD_STATE state_read_thread;

	state_main_thread = MAIN_THREAD_INIT;
	state_h2d_thread = H2D_THREAD_INIT;
	state_read_thread = READ_DISK_THREAD_NEIGHBOR_LF_READ_COMPLETE;
	 
	int dir = 0;
	int while_iter = 0;

	// for result analysis
	std::vector<double> time_end_to_end;
	std::vector<std::pair<size_t, size_t>> reused_per_total;
	std::vector<int> field_mode;
	std::vector<std::pair<int, int>> position_trace;

	/* Main Loop */
	std::mutex mtx;
	std::thread th_h2d(loop_nbrs_h2d, std::ref(LRU), std::ref(LF_window), std::ref(nbrPosition), stream_h2d, std::ref(state_h2d_thread), std::ref(state_main_thread), std::ref(mtx));
	std::thread th_readdisk(loop_read_disk, std::ref(LF_window), std::ref(current_LF_number), std::ref(curPosX), std::ref(state_read_thread), std::ref(state_main_thread));

	while (while_iter < 195) {
		while_iter++;
		prevprevPosX = prevPosX;
		prevprevPosY = prevPosY;
		prevPosX = curPosX;
		prevPosY = curPosY;

#if 0 // AUTO MOVE
		if (dir % 3 == 0)
		{
			// curPosX--;  // DDZ
			// curPosY++;  // DDZ
			curPosX++;  // D
			// curPosY++;  // X 
			// curPosX++;  // WWD
		}
		else
		{
			// curPosX++;  // DDZ
			curPosX++;  // D
			// curPosY++;  // X
			// curPosY--;  // WWD
		}
		dir++;

		printf("\tPosition(%d, %d)\n", curPosX, curPosY);
		state_main_thread = MAIN_THREAD_WAIT;
		getNeighborList(nbrPosition, curPosX, curPosY);
		if (curPosY > 24) {
			break;
		}
#else
		if (state_main_thread != MAIN_THREAD_INIT) {
			if (getKey(curPosX, curPosY) < 0) {
				break;
			}
		}

		state_main_thread = MAIN_THREAD_WAIT;
		getNeighborList(nbrPosition, curPosX, curPosY);
#endif
		sw.Start();
		state_main_thread = MAIN_THREAD_H2D;

		mtx.lock();
		std::pair<size_t, size_t> hitrate = cache_slice(LRU, LF_window, curPosX, curPosY);
		int mode = LRU.synchronize_HashmapOfPtr(LF_window, stream_main, state_read_thread);
		mtx.unlock();

		state_main_thread = MAIN_THREAD_RENDERING;
		rendering << < blocksPerGrid, threadsPerBlock, 0, stream_main >> > (u_synthesized_view, LRU.d_devPtr_hashmap_odd, LRU.d_devPtr_hashmap_even, mode, curPosX, curPosY, g_width, g_height, g_slice_width);
		hipStreamSynchronize(stream_main);
		// main_thread_state = MAIN_THREAD_D2H;
		// hipMemcpyAsync(synthesized_view, u_synthesized_view, g_output_width * g_height * 3, hipMemcpyDeviceToHost, stream_main); 
		state_main_thread = MAIN_THREAD_COMPLETE;

		double stop = sw.Stop();
		time_end_to_end.push_back(stop);
		reused_per_total.push_back(hitrate);
		field_mode.push_back(mode);
		position_trace.push_back(std::make_pair(curPosX, curPosY));
		printf("[%d] %f ms, Cached Slices: %d(Odd), %d(Even)\n", mode, stop, LRU.size(ODD), LRU.size(EVEN));

#if LOGGER==1
		FILE* fv = fopen(("./result/view/[" + std::to_string(g_output_width) + "x" + std::to_string(g_height) + "] " + IntToFormattedString(curPosX) + "_" + IntToFormattedString(curPosY) + ".bgr").c_str(), "wb");
		fwrite(u_synthesized_view, 1, g_output_width * g_height * 3, fv);
		fclose(fv);
#endif
	}
#if LOGGER==1
	FILE* fout_experimental_result = fopen(("./result/ours/" + IntToFormattedString(g_slice_width) + ".log").c_str(), "w");
	fprintf(fout_experimental_result, "mode\tposition\telapsed_time\tresued\ttotal\thitrate\n");
	for (int i = 0; i < time_end_to_end.size(); i++)
	{
		fprintf(fout_experimental_result, "%d\t%d,%d\t%f\t%d\t%d\t%f\n", field_mode.at(i), position_trace.at(i).first, position_trace.at(i).second, time_end_to_end.at(i), reused_per_total.at(i).first, reused_per_total.at(i).second, (double)reused_per_total.at(i).first / (double)reused_per_total.at(i).second);
	}
	fclose(fout_experimental_result);
#endif
	state_main_thread = MAIN_THREAD_TERMINATED;

	/* Destruct */
	if (th_h2d.joinable())
	{
		state_h2d_thread = H2D_THREAD_TERMINATED;
		th_h2d.join();
	}
	if (th_readdisk.joinable())
	{
		state_read_thread = READ_DISK_THREAD_TERMINATED;
		th_readdisk.join();
	}

	for (int i = 0; i < g_LF_window_size; i++) {
		// free_uint8(LF_window.at(i).full_field, "pinned");
		free_uint8(LF_window.at(i).odd_field, "pinned");
		free_uint8(LF_window.at(i).even_field, "pinned");
	}

	free_uint8(u_synthesized_view, "unified");
	hipStreamDestroy(stream_main);
	hipStreamDestroy(stream_h2d);

	return 0;
}